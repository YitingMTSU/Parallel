#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


/*
Name: Yiting Wang
M#: 01360917
Project: CSCI 6330 HW6
*/

# define Nthrds 100 // set the number of the thread

//set initial temperature for matrix
void Initial(double **M,int num_rows, int num_cols, double top,double left, double right, double bottom);

__global__ void calculate(double *dpreV, double *dnewV, double *ddouInfo, int *dintInfo)
{
  int num_rows = dintInfo[0];
  int num_cols = dintInfo[1];
  int step = dintInfo[2];
  int NumThrds = dintInfo[3];
  int rowsearch = dintInfo[4];
  int i,j;
  int tid = threadIdx.x; // get the gpu thread
 
  double eps = ddouInfo[0];
  double Imax_err;

  __shared__ double max_err;
  __shared__ double *err;
  
  err = (double *)malloc(NumThrds*sizeof(double));

  max_err = ddouInfo[1];
  //printf("max_err:%f\n",max_err);

  double loc_err = max_err;
  double temp_err;

  int start_row = (((tid)*rowsearch) > (0) ? ((tid)*rowsearch) : (0));
  int end_row = (((tid+1)*rowsearch + 1) < (num_rows - 1) ? ((tid+1)*rowsearch+1):(num_rows-1));

  if(tid == NumThrds-1)
  {
    end_row = num_rows - 1;
  }
  //printf("rowsearch:%d\n",rowsearch);
  //printf("start: %d end: %d tid: %d\n",start_row, end_row,tid);

  int index;
  while(max_err > eps)
  {
    loc_err = 0.0;
    //get the new value of matrix
    for(i=0;i<(end_row-start_row-1);i++)
    {
      for(j=1;j<num_cols-1;j++)
      {
        index = (start_row+i+1)*num_cols + j;
	dnewV[index] = (dpreV[index-1] + dpreV[index+1] + dpreV[index-num_cols] + dpreV[index+num_cols])/4.0;
	//printf("%f, %f, %f, %f, %f\n",preV[index-1],preV[index+1],preV[index-num_cols],preV[index+num_cols],dnewV[index]);
        temp_err = fabs(dnewV[index] - dpreV[index]);
	//printf("dnewV[%d] = %f, preV[%d] = %f, temp_err:%f, loc_err:%f\n",index,dnewV[index],index,preV[index],temp_err,loc_err);
        if(temp_err>loc_err)
        {
          loc_err = temp_err;
        }
      }
    }

    err[tid] = loc_err;
    //printf("loc_err:%f\n",loc_err);

    __syncthreads();

    //copy back to preV
    for(i=0;i<(end_row-start_row-1);i++)
    {
      for(j=1;j<num_cols-1;j++)
      {
        index = (start_row+1+i)*num_cols + j;
	//printf("index:%d\n",index);
        dpreV[index] = dnewV[index];
      }	
    }
    
    if(tid == NumThrds-1)
    {
 
      Imax_err = err[0];
      //printf("%f,err[0]=%f\n",Imax_err,err[0]);
      for(i=1;i<NumThrds;i++)
      {
        if(Imax_err<err[i])
	Imax_err = err[i];
	//printf("%f\n",Imax_err);
	//printf("%d,%f\n",i,err[i]);
      }
      max_err = Imax_err;
          
      if(step&(step-1))
      {
      }
      else
      {
        if(step>0)
        printf("%6d  %7lf\n",step, max_err);
      }
    }
    step = step+1;
    
    __syncthreads();
}
  if(tid == NumThrds-1)
  {
    printf("%6d  %7lf\n",step, max_err);
  }  	         
}

int main(int argc, char *argv[])
{
  int i;
  double top_temp,left_temp,right_temp,bottom_temp,eps;
  int num_rows, num_cols;
  double **preM; // matrix in host
  double **newM;
  
  double *dpreV; // vector in device
  double *dnewV;
  double *ddouInfo;
  int *dintInfo;
    
  double *preV; // vector in host
  double *newV;
  double *douInfo;
  int *intInfo;
  
  // read the initial number
  num_rows = atoi(argv[1]);
  num_cols = atoi(argv[2]);
  top_temp = atof(argv[3]);
  left_temp = atof(argv[4]);
  right_temp = atof(argv[5]);
  bottom_temp = atof(argv[6]);
  eps = atof(argv[7]);


  //matrix in  the host
  preM = (double **)malloc(num_rows*sizeof(double *));
  newM = (double **)malloc(num_rows*sizeof(double *));

  //malloc in the host
  preV = (double *)malloc(num_rows*num_cols*sizeof(double));
  newV = (double *)malloc(num_rows*num_cols*sizeof(double));
  douInfo = (double *)malloc(2*sizeof(double));
  intInfo = (int *)malloc(5*sizeof(int));
  
  // cuda malloc 
  hipMalloc(&dpreV, sizeof(double)*num_rows*num_cols);
  hipMalloc(&dnewV, sizeof(double)*num_rows*num_cols);
  hipMalloc(&ddouInfo,sizeof(double)*2);
  hipMalloc(&dintInfo,sizeof(int)*5);

  for(i=0;i<num_rows;i++)
  {
    preM[i] = &(preV[i*num_cols]);
    newM[i] = &(newV[i*num_cols]);
  }

  //set initial number for matrix
  Initial(preM,num_rows,num_cols,top_temp,left_temp,right_temp,bottom_temp);

  int rowsearch = ceil((num_rows - 2)*1.0/(Nthrds));

  for(i=0;i<num_cols;i++)
  {
    newM[0][i] = preM[0][i];
    newM[num_rows-1][i] = preM[num_rows-1][i];
  }

  for(i=0;i<num_rows;i++)
  {
    newM[i][0] = preM[i][0];
    newM[i][num_cols-1] = preM[i][num_cols-1];
  }

  //douInfo
  double max_err = 1000.0;
  douInfo[0] = eps;
  douInfo[1] = max_err;

  //intInfo
  int step = 0; 
  intInfo[0] = num_rows;
  intInfo[1] = num_cols;
  intInfo[2] = step;
  intInfo[3] = Nthrds;
  intInfo[4] = rowsearch; 

  // copy from host to device
  hipMemcpy(ddouInfo,douInfo,2*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dintInfo,intInfo,5*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dpreV,preV,num_rows * num_cols * sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dnewV,newV,num_rows * num_cols * sizeof(double),hipMemcpyHostToDevice);

  calculate<<<1,Nthrds>>>(dpreV, dnewV, ddouInfo, dintInfo);

  // copy from device to host
  //cudaMemcpy(douInfo,ddouInfo,2*sizeof(double),cudaMemcpyDeviceToHost);
  //cudaMemcpy(intInfo,dintInfo,5*sizeof(int),cudaMemcpyDeviceToHost);
  
  free(preV);
  free(newV);
  free(douInfo);
  free(intInfo);

  hipFree(dpreV);
  hipFree(dnewV);
  hipFree(ddouInfo);
  hipFree(dintInfo);

  return 0;
}


void Initial(double **M, int num_rows, int num_cols, double top,double left, double right, double bottom)
{
  int i,j,count;
  double sum,average;

  for(i=0;i<num_cols;i++)
  {
    M[0][i] = top;
    M[num_rows-1][i] = bottom;
  }

  for(i=0;i<num_rows-1;i++)
  {
    M[i][0] = left;
    M[i][num_cols-1] = right;
  }
  
  //the sum of the boundary points
  sum = (top)*(num_rows-2) + bottom*(num_rows) + left*(num_cols-1) + right*(num_cols-1);

  //number of points in bound
  count = num_rows*2 + num_cols*2 -4;
  //average of the boundary points
  average = sum*1.0/(count*1.0);
  for (i=1;i<num_rows-1;i++)
  {
    for (j=1;j<num_cols-1;j++)
    {
      M[i][j] = average;
    }
  }
}
											      
